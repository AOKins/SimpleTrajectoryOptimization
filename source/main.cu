#include "hip/hip_runtime.h"
#include <iostream> // For cout
#include "../headers/options.h"
#include "../headers/individual.cuh"
#include "../headers/coords.cuh"
#include "cudaVersion/device.cu" // For callGPU()
#include "cpuVersion/geneticsCPU.cu" // for callCPU()
#include <chrono> // for clock() timing and CLOCKS_PER_SEC

int main(int argc, char *argv[]) { // main.exe input.config <- command to run program with file path to config file being "input.config"
    // Reading config file with path taken from command line
    options * config = new options(argv[1]);
    // Output onto the terminal
    std::cout << *config;
    // Allocate memory for the pool using the config's pop_size
    individual * pool = new individual[config->pop_size];
    float time = 0;
    clock_t start,end;
    // Determine whether to use cuda version of algorithm or cpu only verison
    if (config->useCUDA == true) {
        std::cout << "\n\t\tUSING CUDA\n";
        // Perform version that utilizes CUDA
        start = clock();
        callGPU(pool, config);
        end = clock();
    }
    else {        std::cout << "\n\tNOT\n";

        // Perform CPU only version
        start = clock();
        callCPU(config, pool);
        end = clock();
    }

    time = (float(end) - float(start)) / float(CLOCKS_PER_SEC) * 1000.0;

    std::cout << "Time it took was " << time << " milliseconds\n";
    // Output that the program is finished and deallocate memory
    std::cout <<"Exiting program...";
    delete config;
    delete [] pool;
}
