#include "hip/hip_runtime.h"
#ifndef _PHYSICS_CPP_
#define _PHYSICS_CPP_
// Functions that handle simulating trajectory on device called from kernal

// Determine the acceleration due the atmospher on the obejct due to drag and wind
// Input: constants for physics contants
//        objectPos for current position of the object (currently unused but would be for possibly more complex atmospher simulations)
//        objectVel for current velocity used in deriving the force due to drag/wind that depends on the objects velocity
// Output: returns resulting acceleration in 3D coordinates system
__host__ __device__ data3D calculateAtmosphere( options &constants, data3D objectPos, data3D objectVel) {
    data3D result;
    data3D netSpeed;
    // Get the net/relative speed of the object which depends on wind
    netSpeed.x = (objectVel.x - constants.windcomponents.x);
    netSpeed.y = (objectVel.y - constants.windcomponents.y);
    netSpeed.z = (objectVel.z - constants.windcomponents.z);
    // Get resulting forces on the object using netSpeed
    result.x = 0.5 * constants.obj_dragCoeff * constants.atmosphericDensity * constants.obj_area * netSpeed.x*netSpeed.x / constants.obj_mass;
    result.y = 0.5 * constants.obj_dragCoeff * constants.atmosphericDensity * constants.obj_area * netSpeed.y*netSpeed.y / constants.obj_mass;
    result.z = 0.5 * constants.obj_dragCoeff * constants.atmosphericDensity * constants.obj_area * netSpeed.z*netSpeed.z / constants.obj_mass;
    
    return result;
}

// Determine the acceleration due to gravity on the obejct (currently just constant)
// Input: constants for physics contants
//        objectPos for current position of the object (currently unused but would be used for altitude consideration)
// Output: returns resulting acceleration in 3D coordinates system
__host__ __device__ data3D calculateGravity(options &constants, data3D objectPos) {
    data3D result; // default constructor sets all 3 components initially to 0
    result.z = -constants.gravityAccel;
    return result;
}

// Perform a step of the simulation
// Input: constants - access to physics constants needed and step-size
// Output: object is updated in its simulation by one step size
__host__ __device__ void update(options &constants, individual & object) {
    data3D atm_accel = calculateAtmosphere(constants, object.position, object.velocity);
    data3D grav_accel = calculateGravity(constants, object.position);
    // Get the net acceleration being acted on the object
    double net_accelX, net_accelY, net_accelZ;
    net_accelX = atm_accel.x + grav_accel.x;
    net_accelY = atm_accel.y + grav_accel.y;
    net_accelZ = atm_accel.z + grav_accel.z;
    
    // New values is set to (current value) plus (rate of change) times (step size)
    // Update position
    object.position.x = object.position.x + object.velocity.x * constants.time_stepSize;
    object.position.y = object.position.y + object.velocity.y * constants.time_stepSize;
    object.position.z = object.position.z + object.velocity.z * constants.time_stepSize;
    // Update velocity
    object.velocity.x = object.velocity.x + net_accelX*constants.time_stepSize;
    object.velocity.y = object.velocity.y + net_accelY*constants.time_stepSize;
    object.velocity.z = object.velocity.z + net_accelZ*constants.time_stepSize;
}

// Kernal for performaing simulate across all individuals
// Input: constants - contains constant values needed such as pop_size or physics properties
//        pool - array of individuals to be simulated
//        foundSolution - integer that indicates valid solutions, assumed to be 0
// Output: pool[tid] has cost associated with the parameters
__global__ void simulateGPU(options * constants, individual *pool, int *foundSolution) {
    // Derive id to access appriopriate individual and copy into local memory
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    individual lcl_ind = pool[tid];
    // Also copy constants to local memory
    options lcl_constants = *constants;
    // Reset the initial position (or atleast be sure) to 0,0,0
    lcl_ind.position.x = 0;
    lcl_ind.position.y = 0;
    lcl_ind.position.z = 0;

    // Iterate for each time step until the total triptime is reached/exceded
    for (double c_time = 0; c_time < lcl_ind.time; c_time += lcl_constants.time_stepSize) {
        update(lcl_constants, lcl_ind);
    }
    // Trajectory completed, evaluate cost
    lcl_ind.determineCost(lcl_constants.target_Loc);

    // Store resulting cost to global individual and also set foundSolution to 1 if this individual is a valid solution
    pool[tid].cost = lcl_ind.cost;
    if (lcl_ind.cost < lcl_constants.distance_tol) {
        (*foundSolution) = 1;
    }
}


// Simulate a trajectory using a given object to determine how close it is to the target
// Input: constants - contains needed values such as step size and target location
//        object - the individual that contains the parameters to simulate the trajector (angles, V_nought, and total trip time)
// Output: object contains cost for how close it is to the target and final position starting from 0,0,0 
__host__ void simulate(options constants, individual * object) {
    // Reset the initial position (or atleast be sure) to 0,0,0
    object->position.x = 0;
    object->position.y = 0;
    object->position.z = 0;
    
    // Iterate for each time step until the total triptime is reached
    for (double c_time = 0; c_time <= object->time; c_time += constants.time_stepSize) {
        update(constants, *object);
    }
    // Trajectory completed, evaluate cost
    object->determineCost(constants.target_Loc);
}

#endif
