#include "hip/hip_runtime.h"
#include "../headers/coords.cuh"
#include "../headers/individual.cuh"
#include "../headers/options.h"
#include "physicsCUDA.cu"
#include "genetics.cu"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


// Beating heart
// call randomStart() to fill population with random individuals
// begin loop
// if no solution, 
//      call simulateGPU for this thread
//      perform crossover                          //try to make this pass by reference
__global__ void geneticAlgorithm(individual *pool, options *constants, hiprandState_t *state)
{ 
    // Initially assume we do not have a solution
    bool foundSolution = false;
    int currentGen = 0;
    // Tid value for this thread
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    int leftIndex = (constants->pop_size + tid-1) % constants->pop_size;
    int rightIndex = (constants->pop_size + tid+1) % constants->pop_size;
    // Set pool to contain initially randomly created individuals
    randomStart(pool, *constants, state, tid);

    individual self;
    individual left;
    individual right;
    
    do  {
        simulateGPU(constants, pool, tid);
        self = pool[tid];
        if (self.cost < constants->distance_tol) {
            foundSolution = true;
        }
        __syncthreads();
        if (foundSolution == false) {
            left = pool[leftIndex];
            right = pool[rightIndex];
            __syncthreads();//This is just insureance to make sure that the threads check each other in the right order
            
            // Now checking with neighbors to decide if we should crossover, preference to left (arbitrary)
            if (self.cost > left.cost)
            {
                crossover(&self, &left, *constants, state, tid);
            }
            else if (self.cost > right.cost)
            {
                crossover(&self, &right, *constants, state, tid);            
            }
        }
        //display the best of that generation
        /*
        for (int i = 0; i < currentGen; i++)
        {
            printf("Generation ", i, " best result was: ", pool[tid].cost);
        }
        */
        ++currentGen;
    } while (foundSolution == false && currentGen < constants->max_generations);
    //printf("tid %d : %f", tid, pool[tid].cost);
    //put a statement that states that says you found it
}  


// Kernal caller to manage memory and values needed before calling it
// Input: h_pool - pointer to individual array that holds the individual parameters needing to be computed with
//        h_constants - pointer to options struct that contains the constants needed related to the program
__host__ void callGPU(individual * h_pool, options * h_constants) {
    // Get properties of the gpu to display and also so we could use the maxThreadsPerBlock property
    hipDeviceProp_t * properties = new hipDeviceProp_t;
    hipGetDeviceProperties(properties,0);
    std::cout <<"GPU Properties (" << properties->name << " detected)\n";
    int numThreadsUsed = properties->maxThreadsPerBlock;
    std::cout << "\tThreads used: " << numThreadsUsed << "\n"; 
    // Holds how many blocks to use for the kernal to cover the entire pool, assuming that pop_size is divisible by maxThreadsPerBlock
    int numBlocksUsed = h_constants->pop_size / numThreadsUsed;
    std::cout << "\tBlocks being used: " << numBlocksUsed << "\n";

    // Store the number of bytes the pool array is and use when managing memory for CUDA
    size_t poolMemSize = sizeof(individual)*h_constants->pop_size;

    // Allocate and copy over memory into the device
    individual * d_pool;
    hipMalloc(&d_pool, poolMemSize);
    hipMemcpy(d_pool, h_pool, poolMemSize, hipMemcpyHostToDevice);

    options * d_constants;
    hipMalloc(&d_constants, sizeof(options));
    hipMemcpy(d_constants, h_constants, sizeof(options), hipMemcpyHostToDevice);
    
    // Allocate hiprandState to use for random number generation in CUDA
    hiprandState *d_state;
    hipMalloc(&d_state, h_constants->pop_size);

    // Create and use cudaEvents to sync with and record the outcome
    hipEvent_t begin, end;
    hipEventCreate(&begin);
    hipEventCreate(&end);
    
    hipEventRecord(begin);
    // Initialize the random number generator into state
    initializeRandom<<<numThreadsUsed, numBlocksUsed>>>(d_state, d_constants);
    // Perform the algorithm
    geneticAlgorithm<<<numThreadsUsed, numBlocksUsed>>>(d_pool, d_constants, d_state);
    hipEventRecord(end);

    hipEventSynchronize(end);

    // Copy results of the pool into host memory
    hipMemcpy(h_pool, d_pool, poolMemSize, hipMemcpyDeviceToHost);

    // Free resources from device before ending function
    hipFree(d_constants);
    hipFree(d_pool);
    hipFree(d_state);

    std::sort(h_pool, h_pool + h_constants->pop_size);
    std::cout << "All done!\t" << h_pool[0].cost << "\n";
}