#include "hip/hip_runtime.h"
#include "../headers/coords.cuh"
#include "../headers/individual.cuh"
#include "../headers/options.h"
#include "physicsCUDA.cu"
#include "genetics.cu"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


__global__ void geneticAlgorithm(individual *pool, options *constants, hiprandState_t *state)
{ 
    // Initially assume we do not have a solution
    // Tid value for this thread
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    int leftIndex = (constants->pop_size + tid-1) % constants->pop_size;
    int rightIndex = (constants->pop_size + tid+1) % constants->pop_size;

    // Local holding variables to reduce trips to global memory
    individual self, left, right;
    
    // Copy into local memory
    self = pool[tid];
    left = pool[leftIndex];
    right = pool[rightIndex];
    
    // Now checking with neighbors to decide if we should crossover, preference to left (arbitrary)
    if (self.cost > left.cost)
    {
        crossover(self, left, state, tid);
        pool[tid] = self;
    }
    else if (self.cost > right.cost)
    {
        crossover(self, right, state, tid);
        pool[tid] = self;
    }

}  


// Kernal caller to manage memory and values needed before calling it
// Input: h_pool - pointer to individual array that holds the individual parameters needing to be computed with
//        h_constants - pointer to options struct that contains the constants needed related to the program
__host__ void callGPU(individual * h_pool, options * h_constants) {
    // Get properties of the gpu to display and also so we could use the maxThreadsPerBlock property
    hipDeviceProp_t * properties = new hipDeviceProp_t;
    hipGetDeviceProperties(properties,0);
    std::cout <<"GPU Properties (" << properties->name << " detected)\n";
    int numThreadsUsed = properties->maxThreadsPerBlock;
    std::cout << "\tThreads used: " << numThreadsUsed << "\n"; 
    // Holds how many blocks to use for the kernal to cover the entire pool, assuming that pop_size is divisible by maxThreadsPerBlock
    int numBlocksUsed = h_constants->pop_size / numThreadsUsed;
    std::cout << "\tBlocks being used: " << numBlocksUsed << "\n";

    // Store the number of bytes the pool array is and use when managing memory for CUDA
    size_t poolMemSize = sizeof(individual)*h_constants->pop_size;

    // Allocate and copy over memory into the device
    individual * d_pool;
    hipMalloc(&d_pool, poolMemSize);
    hipMemcpy(d_pool, h_pool, poolMemSize, hipMemcpyHostToDevice);

    options * d_constants;
    hipMalloc(&d_constants, sizeof(options));
    hipMemcpy(d_constants, h_constants, sizeof(options), hipMemcpyHostToDevice);
    
    // Allocate hiprandState to use for random number generation in CUDA
    hiprandState *d_state;
    hipMalloc(&d_state, h_constants->pop_size);

    int * d_foundSolution;
    int * h_foundSolution = new int(0);
    hipMalloc(&d_foundSolution, sizeof(int));
    hipMemcpy(d_constants, h_foundSolution, sizeof(int), hipMemcpyHostToDevice);

    // Create and use cudaEvents to sync with and record the outcome
    hipEvent_t initializeStart, startSimulate;
    hipEvent_t endSimulation, endGenetics;
    hipEventCreate(&initializeStart);
    hipEventCreate(&startSimulate);
    hipEventCreate(&endSimulation);
    hipEventCreate(&endGenetics);
    
    hipEventRecord(initializeStart);
    // Initialize the random number generator into state
    initializeRandom<<<numThreadsUsed, numBlocksUsed>>>(d_pool, d_state, d_constants, d_foundSolution);
    hipEventRecord(startSimulate);
    hipEventSynchronize(startSimulate);
    // At this point all initialization is finished

    int gen_count = 0;
    do {
        // Perform the algorithm
        simulateGPU<<<numThreadsUsed, numBlocksUsed>>>(d_constants, d_pool,  d_foundSolution);
        hipEventRecord(endSimulation);
        hipEventSynchronize(endSimulation);

        // At this point all the simulations are finished including setting costs and found solution determined
        // Copy this boolean to see if a solution was reached
        hipMemcpy(h_foundSolution, d_foundSolution, sizeof(int), hipMemcpyDeviceToHost);
        if (*h_foundSolution == 0) {
            // No solution found yet, create new generation
            geneticAlgorithm<<<numThreadsUsed, numBlocksUsed>>>(d_pool, d_constants, d_state);
            hipEventRecord(endGenetics);
            hipEventSynchronize(endGenetics);
        }
        gen_count++;
        // continue loop until solution found or max generations reached
    } while (*h_foundSolution == 0 && gen_count < h_constants->max_generations);
    std::cout <<"Final " << *h_foundSolution << "-";

    // Copy results of the pool into host memory
    hipMemcpy(h_pool, d_pool, poolMemSize, hipMemcpyDeviceToHost);

    // Free resources from device before ending function
    hipFree(d_constants);
    hipFree(d_pool);
    hipFree(d_state);
    hipFree(d_foundSolution);
    hipEventDestroy(endSimulation);
    delete h_foundSolution;

    // Temp debugging output onto terminal to see rough results of the algorithm
    std::sort(h_pool, h_pool + h_constants->pop_size);
    std::cout << "All done!\t" << h_pool[0].cost << "\n";
}