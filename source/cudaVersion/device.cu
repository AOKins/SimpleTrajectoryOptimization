#include "hip/hip_runtime.h"
#include "../physics.cu"
#include "genetics.cu"
// Includes for cuRAND library to access and use hiprandState to be used in genetic algorithm
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "../../headers/output.h" // for calling output methods


// Kernal caller to manage memory and values needed before calling it
// Input: h_pool - pointer to individual array that holds the individual parameters needing to be computed with
//        h_constants - pointer to options struct that contains the constants needed related to the program
// Output: h_pool may contain individuals with valid solutions to hitting the target
__host__ void callGPU(individual * h_pool, options * h_constants) {
    // Get how many threads and blocks to use
    int numThreadsUsed = h_constants->num_threads_per;
    // Holds how many blocks to use for the kernal to cover the entire pool, assuming that pop_size is equal to num_blocks * numThreads
    int numBlocksUsed = h_constants->num_blocks;

    // Store the number of bytes the pool array is and use when managing memory for CUDA
    size_t poolMemSize = sizeof(individual)*h_constants->pop_size;

    // Allocate and copy over memory into the device
    individual * d_pool;
    hipMalloc(&d_pool, poolMemSize);
    hipMemcpy(d_pool, h_pool, poolMemSize, hipMemcpyHostToDevice);

    individual * d_offset_temp;
    hipMalloc(&d_offset_temp, poolMemSize);

    // Allocate memory for constants object
    options * d_constants;
    hipMalloc(&d_constants, sizeof(options));
    hipMemcpy(d_constants, h_constants, sizeof(options), hipMemcpyHostToDevice);
    
    // Allocate hiprandState to use for random number generation in CUDA
    hiprandState_t *d_state;
    hipMalloc(&d_state, sizeof(hiprandState_t)*h_constants->pop_size);

    // Allocate memory for integer object for determining if solution is found in a thread
    int * d_foundSolution;
    int * h_foundSolution = new int(0);
    hipMalloc(&d_foundSolution, sizeof(int));
    hipMemcpy(d_constants, h_foundSolution, sizeof(int), hipMemcpyHostToDevice);

    // Initialize the random number generator into state
    initializeRandom<<<numBlocksUsed, numThreadsUsed>>>(d_pool, d_state, d_constants, d_foundSolution);
    hipDeviceSynchronize();

    // At this point all initialization is finished
    int gen_count = 0;
    do {
        // Perform the algorithm
        simulateGPU<<<numBlocksUsed, numThreadsUsed>>>(d_constants, d_pool,  d_foundSolution);
        hipDeviceSynchronize();
        // At this point all the simulations are finished including setting costs and found solution determined

        // Copy foundSolution to see if a solution was reached
        hipMemcpy(h_foundSolution, d_foundSolution, sizeof(int), hipMemcpyDeviceToHost);

        if (*h_foundSolution == 0) {            // No solution found yet, create new generation
            geneticAlgorithm<<<numBlocksUsed, numThreadsUsed>>>(d_pool, d_constants, d_state);
            hipDeviceSynchronize();

            // Offset 16 to help diversify the pool, done by calling offsetCopy twice (offset 8 each) to ensure no race condition across all threads
            offsetCopy<<<numBlocksUsed, numThreadsUsed>>>(d_pool, d_offset_temp, d_constants);
            hipDeviceSynchronize();
            offsetCopy<<<numBlocksUsed, numThreadsUsed>>>(d_offset_temp, d_pool, d_constants);
            hipDeviceSynchronize();
        }

        gen_count++; // Increment gen_count for next generation
    } while (*h_foundSolution == 0 && gen_count < h_constants->max_generations); // continue loop until solution found or max generations reached
    // End of algorithm

    // Copy results of the pool into host memory
    hipMemcpy(h_pool, d_pool, poolMemSize, hipMemcpyDeviceToHost);

    // Free resources from device before ending function
    hipFree(d_constants);
    hipFree(d_pool);
    hipFree(d_offset_temp);
    hipFree(d_state);
    hipFree(d_foundSolution);

    // Deallocate host memory
    delete h_foundSolution;
}
